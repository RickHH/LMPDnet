#include "hip/hip_runtime.h"
/**
 * @file lcx_back_tof_lm_cuda.cu
 */

#include<stdio.h>
#include<stdlib.h>
#include<math.h>

extern "C" void lcx_back_tof_lm_cuda(const float *h_xstart,
                                          const float *h_xend,
                                          float **d_img,
                                          const float *h_img_origin,
                                          const float *h_voxsize,
                                          const float *h_p,
                                          float *h_GT,
                                          long long nlors,
                                          const int *h_img_dim,
                                          float tofbin_width,
                                          const float *h_sigma_tof,
                                          const float *h_tofcenter_offset,
                                          float n_sigmas,
                                          const short *h_tof_bin,
                                          unsigned char lor_dependent_sigma_tof,
                                          unsigned char lor_dependent_tofcenter_offset,
                                          int threadsperblock)
{
  // get number of avilable CUDA devices
  int num_devices;
  hipGetDeviceCount(&num_devices);

  // init the dynamic array of device arrays
  float **d_p              = new float * [num_devices];
  float **d_GT              = new float * [num_devices];
  float **d_xstart         = new float * [num_devices];
  float **d_xend           = new float * [num_devices];
  float **d_img_origin     = new float * [num_devices];
  float **d_voxsize        = new float * [num_devices];
  int   **d_img_dim        = new int * [num_devices];

  // init the dynamic arrays of TOF device arrays
  float **d_sigma_tof        = new float * [num_devices];
  float **d_tofcenter_offset = new float * [num_devices];
  short **d_tof_bin          = new short * [num_devices];

  // we split the projections across all CUDA devices
  for (int i_dev = 0; i_dev < num_devices; i_dev++) 
  {
    hipError_t error;  
    int blockspergrid;

    dim3 block(threadsperblock);

    // offset for chunk of projections passed to a device 
    long long dev_offset;
    // number of projections to be calculated on a device
    long long dev_nlors;
    long long proj_bytes_dev;

    long long GT_bytes_dev;

    hipSetDevice(i_dev);
    // () are important in integer division!
    dev_offset = i_dev*(nlors/num_devices);
 
    // calculate the number of projections for a device (last chunck can be a bit bigger) 
    dev_nlors = i_dev == (num_devices - 1) ? (nlors - dev_offset) : (nlors/num_devices);

    // calculate the number of bytes for the projection array on the device
    proj_bytes_dev = dev_nlors*sizeof(float);

    // calculate the number of bytes for the system Matrix GT on the device
    GT_bytes_dev = dev_nlors * h_img_dim[0] * 4 * sizeof(float);

    // calculate the number of blocks needed for every device (chunk)
    blockspergrid = (int)ceil((float)dev_nlors / threadsperblock);
    dim3 grid(blockspergrid);

    // allocate the memory for the array containing the projection on the device
    error = hipMalloc(&d_p[i_dev], proj_bytes_dev);
    if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);}
    hipMemcpyAsync(d_p[i_dev], h_p + dev_offset, proj_bytes_dev, hipMemcpyHostToDevice);

    error = hipMalloc(&d_GT[i_dev], GT_bytes_dev);
    if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);}
    hipMemsetAsync(d_GT[i_dev], 0, GT_bytes_dev);

    error = hipMalloc(&d_xstart[i_dev], 3*proj_bytes_dev);
    if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);}
    hipMemcpyAsync(d_xstart[i_dev], h_xstart + 3*dev_offset, 3*proj_bytes_dev, 
                    hipMemcpyHostToDevice);

    error = hipMalloc(&d_xend[i_dev], 3*proj_bytes_dev);
    if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);}
    hipMemcpyAsync(d_xend[i_dev], h_xend + 3*dev_offset, 3*proj_bytes_dev, 
                    hipMemcpyHostToDevice);
  
    error = hipMalloc(&d_img_origin[i_dev], 3*sizeof(float));
    if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);}
    hipMemcpyAsync(d_img_origin[i_dev], h_img_origin, 3*sizeof(float), 
                    hipMemcpyHostToDevice);

    error = hipMalloc(&d_voxsize[i_dev], 3*sizeof(float));
    if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);}
    hipMemcpyAsync(d_voxsize[i_dev], h_voxsize, 3*sizeof(float), hipMemcpyHostToDevice);

    error = hipMalloc(&d_img_dim[i_dev], 3*sizeof(int));
    if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);}
    hipMemcpyAsync(d_img_dim[i_dev], h_img_dim, 3*sizeof(int), hipMemcpyHostToDevice);


    // send TOF arrays to device
    if (lor_dependent_sigma_tof == 1){
      error = hipMalloc(&d_sigma_tof[i_dev], proj_bytes_dev);
      if (error != hipSuccess){
          printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
          exit(EXIT_FAILURE);}
      hipMemcpyAsync(d_sigma_tof[i_dev], h_sigma_tof + dev_offset, proj_bytes_dev, hipMemcpyHostToDevice);
    }
    else{
      error = hipMalloc(&d_sigma_tof[i_dev], sizeof(float));
      if (error != hipSuccess){
          printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
          exit(EXIT_FAILURE);}
      hipMemcpyAsync(d_sigma_tof[i_dev], h_sigma_tof, sizeof(float), hipMemcpyHostToDevice);
    }

    if (lor_dependent_tofcenter_offset == 1){
      error = hipMalloc(&d_tofcenter_offset[i_dev], proj_bytes_dev);
      if (error != hipSuccess){
          printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
          exit(EXIT_FAILURE);}
      hipMemcpyAsync(d_tofcenter_offset[i_dev], h_tofcenter_offset + dev_offset, proj_bytes_dev, hipMemcpyHostToDevice);
    }
    else{
      error = hipMalloc(&d_tofcenter_offset[i_dev], sizeof(float));
      if (error != hipSuccess){
          printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
          exit(EXIT_FAILURE);}
      hipMemcpyAsync(d_tofcenter_offset[i_dev], h_tofcenter_offset, sizeof(float), hipMemcpyHostToDevice);
    }

    error = hipMalloc(&d_tof_bin[i_dev], dev_nlors*sizeof(short));
    if (error != hipSuccess){
        printf("hipMalloc returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);}
    hipMemcpyAsync(d_tof_bin[i_dev], h_tof_bin + dev_offset, dev_nlors*sizeof(short), hipMemcpyHostToDevice);
    // call the kernel
    lcx_back_tof_lm_cuda_kernel<<<grid,block>>>(d_xstart[i_dev], d_xend[i_dev], d_img[i_dev],
                                                     d_img_origin[i_dev], d_voxsize[i_dev], 
                                                     d_p[i_dev], d_GT[i_dev], dev_nlors, d_img_dim[i_dev],
                                                     tofbin_width, d_sigma_tof[i_dev],
                                                     d_tofcenter_offset[i_dev], n_sigmas, d_tof_bin[i_dev],
                                                     lor_dependent_sigma_tof, lor_dependent_tofcenter_offset);

    // deallocate memory on device
    hipFree(d_p[i_dev]);
    hipFree(d_GT[i_dev]);
    hipFree(d_xstart[i_dev]);
    hipFree(d_xend[i_dev]);
    hipFree(d_img_origin[i_dev]);
    hipFree(d_img_dim[i_dev]);
    hipFree(d_voxsize[i_dev]);

    hipFree(d_sigma_tof[i_dev]);
    hipFree(d_tofcenter_offset[i_dev]);
    hipFree(d_tof_bin[i_dev]);
  }

  hipDeviceSynchronize();
}
